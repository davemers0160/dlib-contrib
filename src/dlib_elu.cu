#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "cuda_dlib.h"
#include "cudnn_dlibapi.h"


namespace dlib 
{ 
    namespace cuda 
    {
	
    // ----------------------------------------------------------------------------------------
/*
        __global__ void _cuda_elu(const float* s, float* d, size_t n, const float* pp)
        {
            const float p = *pp;
            for (auto i : grid_stride_range(0, n))
            {
                if (s[i] > 0)
                    d[i] = s[i];
                else
                    d[i] = p*s[i];
            }
        }

        void elu (
            tensor& dest,
            const tensor& src,
            const tensor& param
        )
        {
            launch_kernel(_cuda_elu, max_jobs(dest.size()), 
                src.device(), dest.device(), src.size(), param.device());
        }
*/
    // ----------------------------------------------------------------------------------------
/*
        __global__ void _cuda_elu_gradient(float* out, const float* s, const float* gi, size_t n, const float* pp)
        {
            const float p = *pp;
            float pgrad = 0;
            for(auto i : grid_stride_range(0, n))
            {
                if (s[i] > 0)
                {
                    out[i] += gi[i];
                }
                else
                {
                    out[i] += p*gi[i];
                    pgrad += gi[i]*s[i];
                }
            }

            // Then do the warp reduce add thing to merge into one output value.
            warp_reduce_atomic_add(*ppgrad, pgrad);
        }

        void elu_gradient (
            tensor& grad,
            const tensor& src,
            const tensor& gradient_input,
            const tensor& param,
            tensor& params_grad 
        )
        {
            params_grad = 0;
            launch_kernel(_cuda_prelu_gradient, max_jobs(grad.size()), 
                grad.device(), src.device(), gradient_input.device(), grad.size(),
                param.device(), params_grad.device());
        }
*/
    // ----------------------------------------------------------------------------------------	
	}
	
}
	